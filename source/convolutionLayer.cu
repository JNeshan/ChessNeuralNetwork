#include "hip/hip_runtime.h"
#include "header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnTensorDescriptor_t inputD, outputD, biasD;
  hipdnnFilterDescriptor_t filterD;
  hipdnnConvolutionDescriptor_t convoD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateTensorDescriptor(&biasD));
    TryCuda(hipdnnCreateConvolutionDescriptor(&convoD));
    TryCuda(hipdnnCreateTensorDescriptor(&gradientD));

  }
  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroyTensorDescriptor(biasD));
    TryCuda(hipdnnDestroyConvolutionDescriptor(convoD));
    TryCuda(hipdnnDestroy(handle));
    TryCuda(hipdnnDestroyTensorDescriptor(gradientD));
  };

  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroyConvolutionDescriptor(convoD));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateConvolutionDescriptor(&convoD));
  }
};

ConvolutionLayer::ConvolutionLayer(const int fC, const int iC, const int fH, const int fW) : bias({fC}, TensorLocation::GPU), filters({fC, iC, fW, fH}, TensorLocation::GPU){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetFilter4dDescriptor(CudaM->filterD, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fC, iC, fH, fW));
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, fC, 1, 1));
  TryCuda(hipdnnSetConvolution2dDescriptor(CudaM->convoD, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));
}

Tensor ConvolutionLayer::forward(const Tensor& T){
  int n = T.dimensions[0], c = T.dimensions[1], h = T.dimensions[2], w = T.dimensions[3]; //dimension variables, reused for memlocations in cudnn calls
  input = Tensor(T); //copies input tensor for backpropagation

  //setting descriptors, calculating output dimensions
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
  TryCuda(hipdnnGetConvolution2dForwardOutputDim(CudaM->convoD, CudaM->inputD, CudaM->filterD, &n, &c, &h, &w));
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
  Tensor output({n, c, h, w}, TensorLocation::GPU); //readies return tensor
  //variables for convolution algorithm and workspace memory
  hipdnnConvolutionFwdAlgoPerf_t potential;
  hipdnnConvolutionFwdAlgo_t algo;
  int algoCount = 0;
  size_t wsSize = 0;
  void* workspace = nullptr;
  float a = 1.0f, b = 1.0f, b2 = 0.0f; //alpha betas
  //finds the best convolution algorithm to use
  TryCuda(hipdnnFindConvolutionForwardAlgorithm(CudaM->handle, CudaM->inputD, CudaM->filterD, CudaM->convoD, 
                                              CudaM->outputD, 1, &algoCount, &potential));
  if(algoCount == 0){ //safety check if none are found
    throw std::runtime_error("cuDNN failed to find convolution");
  }
  algo = potential.algo; 
  //determines the necessary workspace size for convolution
  TryCuda(hipdnnGetConvolutionForwardWorkspaceSize(CudaM->handle, CudaM->inputD, CudaM->filterD, 
                                                  CudaM->convoD, CudaM->outputD, algo, &wsSize));
  if(wsSize > 0){ //allocates necessary workspace space if any
    TryCuda(hipMalloc((void**)&workspace, wsSize));
  }
  //performs convolution
  TryCuda(hipdnnConvolutionForward(CudaM->handle, &a, CudaM->inputD, T.gpuData(), CudaM->filterD, 
                          filters.gpuData(), CudaM->convoD, algo, workspace, 
                          wsSize, &b, CudaM->outputD, output.gpuData()));
  //freeing memory
  if(workspace != nullptr){ //frees the workspace if it was used
    TryCuda(hipFree(workspace));
  }

  //performs bias addition
  TryCuda(hipdnnAddTensor(CudaM->handle, &a, CudaM->biasD, bias.gpuData(), &b2, CudaM->outputD, output.gpuData()));
  return output;
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> ConvolutionLayer::backward(const Tensor& gradient){
  //initializing gradient tensors and descriptor parameters
  iGrad = Tensor(input.dimensions, TensorLocation::GPU);
  bGrad = Tensor(bias.dimensions, TensorLocation::GPU);
  fGrad = Tensor(filters.dimensions, TensorLocation::GPU);

  hipdnnConvolutionBwdFilterAlgoPerf_t potential;
  hipdnnConvolutionBwdFilterAlgo_t algo;
  hipdnnConvolutionBwdDataAlgoPerf_t dataPot;
  hipdnnConvolutionBwdDataAlgo_t dataAlgo;
  int algoCount = 0;
  size_t wsSize = 0, wsSizeTmp;
  void* workspace = nullptr;
  //applies back propagation through the convolutions bias
  TryCuda(hipdnnConvolutionBackwardBias(CudaM->handle, &mx, CudaM->outputD, gradient.gpuData(), &mn, CudaM->outputD, bGrad.gpuData()));
  TryCuda(hipdnnFindConvolutionBackwardDataAlgorithm(CudaM->handle, CudaM->filterD, CudaM->outputD, CudaM->convoD, CudaM->inputD, 1, &algoCount, &dataPot));
  TryCuda(hipdnnFindConvolutionBackwardFilterAlgorithm(CudaM->handle, CudaM->inputD, CudaM->outputD, CudaM->convoD, CudaM->filterD, 1, &algoCount, &potential));
  dataAlgo = dataPot.algo;
  algo = potential.algo;
  TryCuda(hipdnnGetConvolutionBackwardDataWorkspaceSize(CudaM->handle, CudaM->filterD, CudaM->outputD, CudaM->convoD, CudaM->inputD, dataAlgo, &wsSize));
  TryCuda(hipdnnGetConvolutionBackwardFilterWorkspaceSize(CudaM->handle, CudaM->inputD, CudaM->outputD, CudaM->convoD, CudaM->filterD, algo, &wsSizeTmp));
  wsSize = std::max(wsSize, wsSizeTmp);
  if(wsSize > 0){
    TryCuda(hipMalloc((void**)&workspace, wsSize));
  }

  TryCuda(hipdnnConvolutionBackwardData(CudaM->handle, &mx, CudaM->filterD, filters.gpuData(), CudaM->outputD, 
                                      gradient.gpuData(), CudaM->convoD, dataAlgo, workspace, wsSize, &mn, 
                                      CudaM->inputD, iGrad.gpuData()));
  
  TryCuda(hipdnnConvolutionBackwardFilter(CudaM->handle, &mx, CudaM->inputD, input.gpuData(), CudaM->outputD, gradient.gpuData(), 
                                        CudaM->convoD, algo, workspace, wsSize, &mn, CudaM->filterD, fGrad.gpuData()));
  if(workspace != nullptr){
    TryCuda(hipFree(workspace));
    workspace = nullptr;
  }

  return {{&input, &filters, &bias}, {&iGrad, &fGrad, &bGrad}}; //fix output
}

ConvolutionLayer::~ConvolutionLayer(){
  delete CudaM;
}
