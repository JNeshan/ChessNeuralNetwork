#include "header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t* handle;
  
  hipdnnTensorDescriptor_t inputD, outputD, biasD;
  hipdnnFilterDescriptor_t filterD;
  hipdnnConvolutionDescriptor_t convoD;

  CudaMembers(){
    TryCuda(hipdnnCreate(handle));
  }
};

ConvolutionLayer::ConvolutionLayer(const std::vector<int>& dim, int fCount, int fSize) : filterSize(fCount){
  
  std::vector<int> dimensions(1, fCount);
  for(int i = 0; i < dim.size(); i++){
    if(i+2 < dim.size()){ //last two dimensions give the frame
      dimensions.push_back(dim[i]);
    }
    else{
      dimensions.push_back(fSize);
    }
  }

  filters = Tensor(dimensions);
  bias = Tensor({1, filters.size});
  CudaM = CudaMembers();
}

void ConvolutionLayer::forward(Tensor T){
  
}

