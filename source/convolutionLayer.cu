#include "hip/hip_runtime.h"
#include "header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

//think its good


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}



struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnTensorDescriptor_t inputD, outputD, biasD;
  hipdnnFilterDescriptor_t filterD;
  hipdnnConvolutionDescriptor_t convoD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateTensorDescriptor(&biasD));
    TryCuda(hipdnnCreateConvolutionDescriptor(&convoD));
    TryCuda(hipdnnCreateFilterDescriptor(&filterD));
  }
  CudaMembers(const CudaMembers& r){
    this.handle = r.handle;
    this.inputD = r.inputD;
    this.outputD = r.outputD;
    this.biasD = r.biasD;
    this.filterD = r.filterD;
    this.convoD = r.convoD;
  }
  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroyTensorDescriptor(biasD));
    TryCuda(hipdnnDestroyConvolutionDescriptor(convoD));
    TryCuda(hipdnnDestroy(handle));
  };
  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroyConvolutionDescriptor(convoD));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateConvolutionDescriptor(&convoD));
  }
};


ForwardCache::ForwardCache(const Tensor& tensor, CudaMembers& c) : T(tensor){
  this->CudaM = new CudaMembers(c);
}

BackwardCache::BackwardCache() : trainingTensors(0);
void BackwardCache::cachePair(const Tensor& m, const Tensor& grad){
  
}

ConvolutionLayer::ConvolutionLayer(const int fC, const int iC, const int fH, const int fW) : bias({fC}, TensorLocation::GPU), filters({fC, iC, fW, fH}, TensorLocation::GPU){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetFilter4dDescriptor(CudaM->filterD, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fC, iC, fH, fW));
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, fC, 1, 1));
  TryCuda(hipdnnSetConvolution2dDescriptor(CudaM->convoD, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));
}

std::pair<Tensor, std::unique_ptr<ForwardCache>> ConvolutionLayer::forward(const Tensor& T){
  ForwardCache(T);
  int n = T.dimensions[0], c = T.dimensions[1], h = T.dimensions[2], w = T.dimensions[3]; //dimension variables, reused for memlocations in cudnn calls
  input = Tensor(T); //copies input tensor for backpropagation

  //setting descriptors, calculating output dimensions
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w)); //input
  TryCuda(hipdnnGetConvolution2dForwardOutputDim(CudaM->convoD, CudaM->inputD, CudaM->filterD, &n, &c, &h, &w)); //calculates the dimension sizes that the output will have
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w)); //output
  Tensor output({n, c, h, w}, TensorLocation::GPU); //readies return tensor
  //variables for convolution algorithm and workspace memory
  hipdnnConvolutionFwdAlgoPerf_t potential;
  hipdnnConvolutionFwdAlgo_t algo;
  int algoCount = 0;
  size_t wsSize = 0;
  void* workspace = nullptr;
  float a = 1.0f, b = 1.0f, b2 = 0.0f; //alpha betas
  //finds the best convolution algorithm to use
  TryCuda(hipdnnFindConvolutionForwardAlgorithm(CudaM->handle, CudaM->inputD, CudaM->filterD, CudaM->convoD, 
                                              CudaM->outputD, 1, &algoCount, &potential));
  if(algoCount == 0){ //safety check if none are found
    throw std::runtime_error("cuDNN failed to find convolution");
  }
  algo = potential.algo; 
  //determines the necessary workspace size for convolution
  TryCuda(hipdnnGetConvolutionForwardWorkspaceSize(CudaM->handle, CudaM->inputD, CudaM->filterD, 
                                                  CudaM->convoD, CudaM->outputD, algo, &wsSize));
  if(wsSize > 0){ //allocates necessary workspace space if any
    TryCuda(hipMalloc((void**)&workspace, wsSize));
  }
  //performs convolution
  TryCuda(hipdnnConvolutionForward(CudaM->handle, &mx, CudaM->inputD, T.gpuData(), CudaM->filterD, 
                          filters.gpuData(), CudaM->convoD, algo, workspace, 
                          wsSize, &mn, CudaM->outputD, output.gpuData()));
  //freeing memory
  if(workspace != nullptr){ //frees the workspace if it was used
    TryCuda(hipFree(workspace));
  }

  //performs bias addition
  TryCuda(hipdnnAddTensor(CudaM->handle, &a, CudaM->biasD, bias.gpuData(), &b2, CudaM->outputD, output.gpuData()));
  return output;
}
std::pair<Tensor, std::unique_ptr<BackwardCache>> ConvolutionLayer::backward(const Tensor& gradient, const ForwardCache& fCache){
  //initializing gradient tensors and descriptor parameters
  Tensor iGrad(input.dimensions, TensorLocation::GPU, input.n);
  bGrad = Tensor(bias.dimensions, TensorLocation::GPU, bias.n);
  fGrad = Tensor(filters.dimensions, TensorLocation::GPU, filters.n);
  //bad naming, first two for filter, last two for the returned gradient
  hipdnnConvolutionBwdFilterAlgoPerf_t potential;
  hipdnnConvolutionBwdFilterAlgo_t algo;
  hipdnnConvolutionBwdDataAlgoPerf_t dataPot;
  hipdnnConvolutionBwdDataAlgo_t dataAlgo;

  int algoCount = 0;
  size_t wsSize = 0, wsSizeTmp;
  void* workspace = nullptr;
  //applies back propagation through the convolutions bias
  TryCuda(hipdnnConvolutionBackwardBias(CudaM->handle, &mx, CudaM->outputD, gradient.gpuData(), &mn, CudaM->outputD, bGrad.gpuData()));
  TryCuda(hipdnnFindConvolutionBackwardDataAlgorithm(CudaM->handle, CudaM->filterD, CudaM->outputD, CudaM->convoD, CudaM->inputD, 1, &algoCount, &dataPot));
  TryCuda(hipdnnFindConvolutionBackwardFilterAlgorithm(CudaM->handle, CudaM->inputD, CudaM->outputD, CudaM->convoD, CudaM->filterD, 1, &algoCount, &potential));
  dataAlgo = dataPot.algo;
  algo = potential.algo;
  TryCuda(hipdnnGetConvolutionBackwardDataWorkspaceSize(CudaM->handle, CudaM->filterD, CudaM->outputD, CudaM->convoD, CudaM->inputD, dataAlgo, &wsSize));
  TryCuda(hipdnnGetConvolutionBackwardFilterWorkspaceSize(CudaM->handle, CudaM->inputD, CudaM->outputD, CudaM->convoD, CudaM->filterD, algo, &wsSizeTmp));
  wsSize = std::max(wsSize, wsSizeTmp);
  if(wsSize > 0){ //allocates space for workspace if necessary, uses the higher space requirement so both can use the same allocation on their turn
    TryCuda(hipMalloc((void**)&workspace, wsSize));
  }

  TryCuda(hipdnnConvolutionBackwardData(CudaM->handle, &mx, CudaM->filterD, filters.gpuData(), CudaM->outputD, 
                                      gradient.gpuData(), CudaM->convoD, dataAlgo, workspace, wsSize, &mn, 
                                      CudaM->inputD, iGrad.gpuData()));
  
  TryCuda(hipdnnConvolutionBackwardFilter(CudaM->handle, &mx, CudaM->inputD, input.gpuData(), CudaM->outputD, gradient.gpuData(), 
                                        CudaM->convoD, algo, workspace, wsSize, &mn, CudaM->filterD, fGrad.gpuData()));
  if(workspace != nullptr){
    TryCuda(hipFree(workspace));
    workspace = nullptr;
  }
  CudaM->resetTemp();
  return iGrad;
}

ConvolutionLayer::~ConvolutionLayer(){
  delete CudaM;
}
