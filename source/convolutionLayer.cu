#include "hip/hip_runtime.h"
#include "../header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

//think its good - this is not true at all

thread_local hipdnnHandle_t Layer::nnHandle{};
thread_local hipblasHandle_t Layer::blasHandle{};

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void tensorDesc(hipdnnTensorDescriptor_t& desc, const Tensor& T){
  TryCuda(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], T.dimensions[1], T.dimensions[2], T.dimensions[3]));
}

/*
bias and filter members
hipdnnTensorDescriptor_t inputD, outputD; hipdnnConvolutionDescriptor_t convoD;
hipdnnFilterDescriptor_t filterD; hipdnnTensorDescriptor_t biasD;
*/


ConvolutionLayer::ConvolutionLayer(const int fC, const int iC, const int fH, const int fW) : bias({fC}, TensorLocation::GPU), filters({fC, iC, fH, fW}, TensorLocation::GPU){
  TryCuda(hipdnnCreateTensorDescriptor(&inputD));
  TryCuda(hipdnnCreateTensorDescriptor(&outputD));
  TryCuda(hipdnnCreateTensorDescriptor(&biasD));
  TryCuda(hipdnnCreateFilterDescriptor(&filterD));
  TryCuda(hipdnnCreateConvolutionDescriptor(&convoD));
  TryCuda(hipdnnSetFilter4dDescriptor(filterD, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fC, iC, fH, fW));
  TryCuda(hipdnnSetTensor4dDescriptor(biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, fC, 1, 1));
}

Tensor ConvolutionLayer::forward(const Tensor& T, bool train){

  int n = T.dimensions[0], c = T.dimensions[1], h = T.dimensions[2], w = T.dimensions[3];
  input = Tensor(T);

  //setting descriptors, calculating output dimensions
  TryCuda(hipdnnSetTensor4dDescriptor(inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w)); //input
  TryCuda(hipdnnGetConvolution2dForwardOutputDim(convoD, inputD, filterD, &n, &c, &h, &w)); //calculates the dimension sizes that the output will have
  TryCuda(hipdnnSetTensor4dDescriptor(outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w)); //output
  Tensor output({n, c, h, w}, TensorLocation::GPU); //readies return tensor
  //variables for convolution algorithm and workspace memory
  hipdnnConvolutionFwdAlgoPerf_t potential;
  hipdnnConvolutionFwdAlgo_t algo;
  int algoCount = 0;
  size_t wsSize = 0;
  void* workspace = nullptr;
  float a = 1.0f, b = 1.0f, b2 = 0.0f; //alpha betas
  //finds the best convolution algorithm to use
  TryCuda(hipdnnFindConvolutionForwardAlgorithm(nnHandle, inputD, filterD, convoD, 
                                              outputD, 1, &algoCount, &potential));
  if(algoCount == 0){ //safety check if none are found
    throw std::runtime_error("cuDNN failed to find convolution");
  }
  algo = potential.algo; 
  //determines the necessary workspace size for convolution
  TryCuda(hipdnnGetConvolutionForwardWorkspaceSize(nnHandle, inputD, filterD, 
                                                  convoD, outputD, algo, &wsSize));
  if(wsSize > 0){ //allocates necessary workspace space if any
    TryCuda(hipMalloc((void**)&workspace, wsSize));
  }
  //performs convolution
  TryCuda(hipdnnConvolutionForward(nnHandle, &mx, inputD, T.gpuData(), filterD, 
                          filters.gpuData(), convoD, algo, workspace, 
                          wsSize, &mn, outputD, output.gpuData()));
  //freeing memory
  if(workspace != nullptr){ //frees the workspace if it was used
    TryCuda(hipFree(workspace));
  }

  //performs bias addition
  TryCuda(hipdnnAddTensor(nnHandle, &a, biasD, bias.gpuData(), &b2, outputD, output.gpuData()));  
  return output;
}

Tensor ConvolutionLayer::backward(const Tensor& gradient){
  //initializing gradient tensors and descriptor parameters
  
  iGrad = Tensor(input.dimensions, TensorLocation::GPU, input.n);
  bGrad = Tensor(bias.dimensions, TensorLocation::GPU, bias.n);
  fGrad = Tensor(filters.dimensions, TensorLocation::GPU, filters.n);
  //bad naming, first two for filter, last two for the returned gradient
  hipdnnConvolutionBwdFilterAlgoPerf_t potential;
  hipdnnConvolutionBwdFilterAlgo_t algo;
  hipdnnConvolutionBwdDataAlgoPerf_t dataPot;
  hipdnnConvolutionBwdDataAlgo_t dataAlgo;

  int algoCount = 0;
  size_t wsSize = 0, wsSizeTmp;
  void* workspace = nullptr;
  //applies back propagation through the convolutions bias
  TryCuda(hipdnnConvolutionBackwardBias(nnHandle, &mx, outputD, gradient.gpuData(), &mn, biasD, bGrad.gpuData()));
  TryCuda(hipdnnFindConvolutionBackwardDataAlgorithm(nnHandle, filterD, outputD, convoD, inputD, 1, &algoCount, &dataPot));
  TryCuda(hipdnnFindConvolutionBackwardFilterAlgorithm(nnHandle, inputD, outputD, convoD, filterD, 1, &algoCount, &potential));
  dataAlgo = dataPot.algo;
  algo = potential.algo;
  TryCuda(hipdnnGetConvolutionBackwardDataWorkspaceSize(nnHandle, filterD, outputD, convoD, inputD, dataAlgo, &wsSize));
  TryCuda(hipdnnGetConvolutionBackwardFilterWorkspaceSize(nnHandle, inputD, outputD, convoD, filterD, algo, &wsSizeTmp));
  wsSize = std::max(wsSize, wsSizeTmp);
  if(wsSize > 0){ //allocates space for workspace if necessary, uses the higher space requirement so both can use the same allocation on their turn
    TryCuda(hipMalloc((void**)&workspace, wsSize));
  }

  TryCuda(hipdnnConvolutionBackwardData(nnHandle, &mx, filterD, filters.gpuData(), outputD, 
                                      gradient.gpuData(), convoD, dataAlgo, workspace, wsSize, &mn, 
                                      inputD, iGrad.gpuData()));
  
  TryCuda(hipdnnConvolutionBackwardFilter(nnHandle, &mx, inputD, input.gpuData(), outputD, gradient.gpuData(), 
                                        convoD, algo, workspace, wsSize, &mn, filterD, fGrad.gpuData()));
  if(workspace != nullptr){
    TryCuda(hipFree(workspace));
    workspace = nullptr;
  }
  return iGrad;
}

ConvolutionLayer::~ConvolutionLayer(){
  TryCuda(hipdnnDestroyFilterDescriptor(filterD));
  TryCuda(hipdnnDestroyTensorDescriptor(inputD));
  TryCuda(hipdnnDestroyTensorDescriptor(outputD));
  TryCuda(hipdnnDestroyTensorDescriptor(biasD));
  TryCuda(hipdnnDestroyConvolutionDescriptor(convoD));
}
