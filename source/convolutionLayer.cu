#include "hip/hip_runtime.h"
#include "header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnTensorDescriptor_t inputD, outputD, biasD;
  hipdnnFilterDescriptor_t filterD;
  hipdnnConvolutionDescriptor_t convoD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateTensorDescriptor(&biasD));
    TryCuda(hipdnnCreateConvolutionDescriptor(&convoD));
  }
  ~CudaMembers(){
  };
};

ConvolutionLayer::ConvolutionLayer(const int fC, const int iC, const int fH, const int fW) : bias({fC}, TensorLocation::GPU), filters({fC, iC, fW, fH}, TensorLocation::GPU) {
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetFilter4dDescriptor(CudaM->filterD, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fC, iC, fH, fW));
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, fC, 1, 1));
  TryCuda(hipdnnSetConvolution2dDescriptor(CudaM->convoD, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));
}

Tensor ConvolutionLayer::forward(const Tensor& T){
  int n = T.dimensions[0], c = (T.dimensions[1] / ((int)pow(8, 4 - T.dimensions.size()))), 
  h = 8, w = 8;

  input = Tensor(T);
  if(input.dimensions.size() != 4){
    throw("Convolution bad input");
  }

  //setting descriptors, calculating output dimensions
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
  TryCuda(hipdnnGetConvolution2dForwardOutputDim(CudaM->convoD, CudaM->inputD, CudaM->filterD, &n, &c, &h, &w));
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
  Tensor output({n, c, h, w}, TensorLocation::GPU);
  //variables for convolution algorithm and workspace memory
  hipdnnConvolutionFwdAlgoPerf_t potential;
  hipdnnConvolutionFwdAlgo_t algo;
  int algoCount = 0;
  size_t wsSize = 0;
  void* workspace = nullptr;
  float a = 1.0f, b = 1.0f, b2 = 0.0f; //alpha betas
  
  TryCuda(hipdnnFindConvolutionForwardAlgorithm(CudaM->handle, CudaM->inputD, CudaM->filterD, CudaM->convoD, 
                                              CudaM->outputD, 1, &algoCount, &potential));
  if(algoCount == 0){
    throw std::runtime_error("cuDNN failed to find convolution");
  }
  algo = potential.algo;

  TryCuda(hipdnnGetConvolutionForwardWorkspaceSize(CudaM->handle, CudaM->inputD, CudaM->filterD, 
                                                  CudaM->convoD, CudaM->outputD, algo, &wsSize));
  if(wsSize > 0){
    TryCuda(hipMalloc((void**)&workspace, wsSize));
  }
  //performs convolution
  TryCuda(hipdnnConvolutionForward(CudaM->handle, &a, CudaM->inputD, T.gpuData(), CudaM->filterD, 
                          filters.gpuData(), CudaM->convoD, algo, workspace, 
                          wsSize, &b, CudaM->outputD, output.gpuData()));
  //freeing memory
  if(workspace != nullptr){
    TryCuda(hipFree(workspace));
  }

  //performs bias addition
  TryCuda(hipdnnAddTensor(CudaM->handle, &a, CudaM->biasD, bias.gpuData(), &b2, CudaM->outputD, output.gpuData()));
  return output;
}

ConvolutionLayer::~ConvolutionLayer(){
  delete CudaM;
}
