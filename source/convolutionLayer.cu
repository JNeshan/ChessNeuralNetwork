#include "header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t* handle;
  
  hipdnnTensorDescriptor_t inputD, outputD, biasD;
  hipdnnFilterDescriptor_t filterD;
  hipdnnConvolutionDescriptor_t convoD;

  CudaMembers(){
    TryCuda(hipdnnCreate(handle));
  }
};

ConvolutionLayer::ConvolutionLayer(std::vector<int>& dim, int fCount, int fSize) : filterSize(fCount){
  filters = Tensor(dim);
  bias = Tensor({1, filters.size});
  CudaM = CudaMembers();
}

void ConvolutionLayer::forward(Tensor T){
  hipdnnHandle_t* hol = CudaM.handle;
}

