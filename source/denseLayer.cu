#include "header/denseLayer.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipblasStatus_t err){
  if(err != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "cuBLAS Error in %s at line %d: %s\n", __FILE__, __LINE__, cublasGetStatusString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipblasHandle_t handle;
  hipdnnHandle_t nHandle;
  hipdnnTensorDescriptor_t outD, biasD;

  CudaMembers(){
    hipblasCreate(&handle);
    hipdnnCreate(&nHandle);
    hipdnnCreateTensorDescriptor(&outD);
    hipdnnCreateTensorDescriptor(&biasD);
  }

  ~CudaMembers(){}
};

DenseLayer::DenseLayer(const int f, const int n) : weight({f, n}, TensorLocation::GPU), bias({1, n}, TensorLocation::GPU){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, n));
}


Tensor DenseLayer::forward(const Tensor& T){
  if(T.dimensions[3] != weight.dimensions[2]){
    throw("Weight and input tensor incorrect dimensions for multiplication");
  }

  Tensor output({T.dimensions[0], weight.dimensions[1]}, TensorLocation::GPU);
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->outD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, output.dimensions[2], output.dimensions[3]));
  TryCuda(hipblasSgemm(CudaM->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, weight.dimensions[3], T.dimensions[2], 
                        T.dimensions[3], &mx, weight.gpuData(), weight.dimensions[3], T.gpuData(), 
                        T.dimensions[3], &mn, output.gpuData(), output.dimensions[3]));
  TryCuda(hipdnnAddTensor(CudaM->nHandle, &mx, CudaM->outD, output.gpuData(), &mx, CudaM->biasD, bias.gpuData()));
  return output;
}

DenseLayer::~DenseLayer(){
  delete CudaM;
}