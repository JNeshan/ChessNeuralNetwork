#include "header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}



struct CudaMembers{
  hipblasStatus_t stat;

};