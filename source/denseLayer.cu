#include "hip/hip_runtime.h"
#include "header/denseLayer.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipblasStatus_t err){
  if(err != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "cuBLAS Error in %s at line %d: %s\n", __FILE__, __LINE__, cublasGetStatusString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipblasHandle_t handle;
  hipdnnHandle_t nHandle;
  hipdnnTensorDescriptor_t outputD, biasD, gradientD;

  CudaMembers(){
    hipblasCreate(&handle);
    hipdnnCreate(&nHandle);
    hipdnnCreateTensorDescriptor(&outputD);
    hipdnnCreateTensorDescriptor(&biasD);
    TryCuda(hipdnnCreateTensorDescriptor(&gradientD));

  }

  ~CudaMembers(){
    TryCuda(hipblasDestroy(handle));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroyTensorDescriptor(biasD));
    TryCuda(hipdnnDestroy(nHandle));
    TryCuda(hipdnnDestroyTensorDescriptor(gradientD));
  };

  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
  }
};
//uses
__global__ void bGradKernel(const float* grad, float* out, const int m, const int n){
  extern __shared__ float shared[]; //initializes space for shared memory
  const int colIdx = blockIdx.x; //column index the thread block works on
  int thId = threadIdx.x; //threads relative id
  shared[thId] = 0.0f; //initializing shared memory values
  for(int row = thId; row < m; row++){ //adding gradient values to shared memory
    shared[thId] += grad[row * n + colIdx];
  }
  __syncthreads();

  for(int str = blockDim.x / 2; str > 0; str >>= 1){ //summing each column, applying half to the other half each time
    if(thId < str){ //indicates which threads are still allowed
      shared[thId] += shared[thId + str];
    }
    __syncthreads(); //waits to sync each iteration
  }

  if(thId == 0){ //one thread sets the sum of its column
    out[colIdx] = shared[0]; 
  }
}

DenseLayer::DenseLayer(const int f, const int n) : weight({f, n}, TensorLocation::GPU), bias({1, n}, TensorLocation::GPU){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, 1, 1, 1));
}


Tensor DenseLayer::forward(const Tensor& T){
  if(T.dimensions[1] != weight.dimensions[0]){
    throw("Weight and input tensor dimensions incompatible for multiplication");
  }
  input = T;
  Tensor output({T.dimensions[0], weight.dimensions[1]}, TensorLocation::GPU);
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output.dimensions[0], output.dimensions[1], 1, 1));
  TryCuda(hipblasSgemm(CudaM->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, weight.dimensions[1], input.dimensions[0], input.dimensions[1],
                       &mx, weight.gpuData(), weight.dimensions[1], input.gpuData(), input.dimensions[1], &mn, output.gpuData(), output.dimensions[2]));
  TryCuda(hipdnnAddTensor(CudaM->nHandle, &mx, CudaM->outputD, output.gpuData(), &mx, CudaM->biasD, bias.gpuData()));
  return output;
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> DenseLayer::backward(const Tensor& gradient){
  iGrad = Tensor (input.dimensions, TensorLocation::GPU);
  wGrad = Tensor (weight.dimensions, TensorLocation::GPU);
  bGrad = Tensor (bias.dimensions, TensorLocation::GPU);

  TryCuda(hipblasSgemm(CudaM->handle, HIPBLAS_OP_T, HIPBLAS_OP_N, gradient.dimensions[0], weight.dimensions[0], 
                        gradient.dimensions[1], &mx, gradient.gpuData(), gradient.dimensions[1], weight.gpuData(), 
                        weight.dimensions[1], &mn, iGrad.gpuData(), iGrad.dimensions[1]));
  //calculates weight gradient by 
  TryCuda(hipblasSgemm(CudaM->handle, HIPBLAS_OP_N, HIPBLAS_OP_T, input.dimensions[1], gradient.dimensions[1], 
                        input.dimensions[0], &mx, input.gpuData(), input.dimensions[1], gradient.gpuData(), 
                        gradient.dimensions[1], &mn, wGrad.gpuData(), wGrad.dimensions[1]));
  //TryCuda()
  int thCount = 256; //threads per block
  while(thCount < input.dimensions[0]){
    thCount *= 2;
  }
  dim3 gridDim(gradient.dimensions[1]);
  dim3 blockDim(thCount); //one dimensional block of th threads
  size_t shrMemSize = thCount * sizeof(float); //size in memory of each block
  bGradKernel<<<gridDim, blockDim, shrMemSize>>>(gradient.gpuData(), bGrad.gpuData(), gradient.dimensions[0], gradient.dimensions[1]);
  CudaM->resetTemp();
  return {{&input, &weight, &bias}, {&iGrad, &wGrad, &bGrad}};
}

DenseLayer::~DenseLayer(){
  delete CudaM;
}