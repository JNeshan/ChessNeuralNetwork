#include "../header/layer.h"

hipdnnHandle_t nnCreate(){
  hipdnnHandle_t handle;
  hipdnnCreate(&handle);
  return handle;
}

hipblasHandle_t blasCreate(){
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  return handle;
}

thread_local hipdnnHandle_t Layer::nnHandle = nnCreate();
thread_local hipblasHandle_t Layer::blasHandle = blasCreate();

Layer::Layer(){
}

Layer::~Layer(){
  
}