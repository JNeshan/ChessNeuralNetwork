#include "header/reluLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}


struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnActivationDescriptor_t reLU;
  hipdnnTensorDescriptor_t tensorD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
    TryCuda(hipdnnCreateActivationDescriptor(&reLU));

  }
  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
    TryCuda(hipdnnDestroyActivationDescriptor(reLU));
    TryCuda(hipdnnDestroy(handle));
  }
  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
    TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  }
};

ReLULayer::ReLULayer(){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetActivationDescriptor(CudaM->reLU, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 2));

}

std::pair<Tensor, std::unique_ptr<ForwardCache>> ReLULayer::forward(const Tensor& T){
  Tensor output(T.dimensions, TensorLocation::GPU, T.size);
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.size, 1, 1, 1));
  TryCuda(hipdnnActivationForward(CudaM->handle, CudaM->reLU, &alpha, CudaM->tensorD, T.gpuData(), &beta, CudaM->tensorD, output.gpuData()));
  return output;
}

std::pair<Tensor, std::unique_ptr<BackwardCache>> ReLULayer::backward(const Tensor& gradient, const ForwardCache& fCache){
  Tensor iGrad(input.dimensions, TensorLocation::GPU, input.n);
  TryCuda(hipdnnActivationBackward(CudaM->handle, CudaM->reLU, &mx, CudaM->tensorD, gradient.gpuData(), CudaM->tensorD, gradient.gpuData(), CudaM->tensorD, input.gpuData(), &mn, CudaM->tensorD, iGrad.gpuData()));  
  BackwardCache back();
  std::pair<std::vector<Tensor*>, std::vector<Tensor*>>
  return iGrad;
}

ReLULayer::~ReLULayer(){
  delete CudaM;
}