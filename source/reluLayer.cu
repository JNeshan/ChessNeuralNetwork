#include "header/reluLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}


struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnActivationDescriptor_t reLU;
  hipdnnTensorDescriptor_t inputD, outputD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateActivationDescriptor(&reLU));

  }
  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroyActivationDescriptor(reLU));
    TryCuda(hipdnnDestroy(handle));
  };
  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
  }
};

ReLULayer::ReLULayer(){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetActivationDescriptor(CudaM->reLU, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 2));

}

Tensor ReLULayer::forward(const Tensor& T){
  Tensor output(T.dimensions, TensorLocation::GPU);
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], T.dimensions[1], T.dimensions[2], T.dimensions[3]));
  TryCuda(hipdnnActivationForward(CudaM->handle, CudaM->reLU, &alpha, CudaM->inputD, T.gpuData(), &beta, CudaM->outputD, output.gpuData()));
  CudaM->resetTemp();
  return output;
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> ReLULayer::backward(const Tensor& gradient){
  iGrad = Tensor(input.dimensions, TensorLocation::GPU);
  TryCuda(hipdnnActivationBackward(CudaM->handle, CudaM->reLU, &mx, CudaM->outputD, gradient.gpuData(), CudaM->outputD, gradient.gpuData(), CudaM->inputD, input.gpuData(), &mn, CudaM->inputD, iGrad.gpuData()));
  
  CudaM->resetTemp();
  return {{&input}, {&iGrad}};
}

ReLULayer::~ReLULayer(){
  delete CudaM;
}