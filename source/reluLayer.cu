#include "header/reluLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

void ReLULayer::forward(Tensor T){
  
  
}

void ReLULayer::backward(){
  return;
}
