#include "header/reluLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}


struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnActivationDescriptor_t reLU;
  hipdnnTensorDescriptor_t inputD;
  hipdnnTensorDescriptor_t outputD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateActivationDescriptor(&reLU));
  }
  ~CudaMembers(){
  }
};

ReLULayer::ReLULayer(){
  CudaM = new CudaMembers();
}

Tensor ReLULayer::forward(const Tensor& T){
  Tensor output(T.dimensions, TensorLocation::GPU);
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], T.dimensions[1], T.dimensions[2], T.dimensions[3]));
  TryCuda(hipdnnSetActivationDescriptor(CudaM->reLU, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 2));
  TryCuda(hipdnnActivationForward(CudaM->handle, CudaM->reLU, &alpha, CudaM->inputD, T.gpuData(), &beta, CudaM->outputD, output.gpuData()));
  return output;
}

void ReLULayer::backward(){
  return;
}

ReLULayer::~ReLULayer(){
  delete CudaM;
}