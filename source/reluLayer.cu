#include "../header/reluLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>

thread_local hipdnnHandle_t Layer::nnHandle{};
thread_local hipblasHandle_t Layer::blasHandle{};

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

ReLULayer::ReLULayer(){
  TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  TryCuda(hipdnnCreateActivationDescriptor(&reLU));  
  TryCuda(hipdnnSetActivationDescriptor(reLU, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 2));

}

Tensor ReLULayer::forward(const Tensor& T, bool train){
  Tensor output(T.dimensions, TensorLocation::GPU, T.size);
  TryCuda(hipdnnSetTensor4dDescriptor(tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.size, 1, 1, 1));
  TryCuda(hipdnnActivationForward(nnHandle, reLU, &mx, tensorD, T.gpuData(), &mn, tensorD, output.gpuData()));
  return output;
}

Tensor ReLULayer::backward(const Tensor& gradient){
  Tensor iGrad(input.dimensions, TensorLocation::GPU, input.n);
  TryCuda(hipdnnActivationBackward(nnHandle, reLU, &mx, tensorD, gradient.gpuData(), tensorD, gradient.gpuData(), tensorD, input.gpuData(), &mn, tensorD, iGrad.gpuData()));  
  return iGrad;
}

ReLULayer::~ReLULayer(){
  TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
  TryCuda(hipdnnDestroyActivationDescriptor(reLU));
}