#include "header/softmaxLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnTensorDescriptor_t tensorD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  }

  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
    TryCuda(hipdnnDestroy(handle));
  };
};

SoftmaxLayer::SoftmaxLayer() : outFeat(4672){
  CudaM = new CudaMembers();
}

SoftmaxLayer::~SoftmaxLayer(){
  delete CudaM;
}

std::pair<Tensor, std::unique_ptr<ForwardCache>> SoftmaxLayer::forward(const Tensor& T){
  if(T.n != 2){ //input must be 2 dimensional
    throw("Softmax input invalid n = " + std::to_string(T.n));
  }
  Tensor({T.dimensions[0], outFeat}, TensorLocation::GPU);
  output = Tensor({T.dimensions[0], outFeat}, TensorLocation::GPU); //storing output for back
  if(output.size != T.size){ //check to ensure the matrices are the same size (also means 2nd dimensions are equal)
    throw("Bad softmax input"); 
  }
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], T.dimensions[1], 1, 1)); //input descriptor
  TryCuda(hipdnnSoftmaxForward(CudaM->handle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, CudaM->tensorD, T.gpuData(), &mn, CudaM->tensorD, output.gpuData()));
  return output;
}

std::pair<Tensor, std::unique_ptr<BackwardCache>> SoftmaxLayer::backward(const Tensor& gradient, const ForwardCache& fCache){
  if(gradient.n != 2 || gradient.size < output.size){
    throw("Softmax recieved bad gradient or recorded faulty output");
  }
  Tensor iGrad(output.dimensions, TensorLocation::GPU, output.n);
  TryCuda(hipdnnSoftmaxBackward(CudaM->handle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, CudaM->tensorD, 
                              output.gpuData(), CudaM->tensorD, gradient.gpuData(), &mn, CudaM->tensorD, iGrad.gpuData()));
  CudaM->resetTemp();
  return iGrad;
}
