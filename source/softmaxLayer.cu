#include "header/softmaxLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnTensorDescriptor_t inputD, outputD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
  }

  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
  }

  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroy(handle));
  };
};

SoftmaxLayer::SoftmaxLayer() : outFeat(4672){
  CudaM = new CudaMembers();
}

SoftmaxLayer::~SoftmaxLayer(){
  delete CudaM;
}

Tensor SoftmaxLayer::forward(const Tensor& T){
  output = Tensor({1, 1, input.dimensions[0], outFeat}, TensorLocation::GPU);
  if(output.size != T.size){
    throw("Bad softmax input");
  }
  input = Tensor(T);
  iGrad = Tensor(input.dimensions, TensorLocation::GPU);
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input.dimensions[0], outFeat, 1, 1));
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input.dimensions[0], input.size / input.dimensions[0], 1, 1));
  TryCuda(hipdnnSoftmaxForward(CudaM->handle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, CudaM->inputD, input.gpuData(), &mn, CudaM->outputD, output.gpuData()));
  return output;
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> SoftmaxLayer::backward(const Tensor& gradient){
  
  Tensor grad(input.dimensions, TensorLocation::GPU);
  TryCuda(hipdnnSoftmaxBackward(CudaM->handle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, CudaM->outputD, 
                              output.gpuData(), CudaM->outputD, gradient.gpuData(), &mn, CudaM->inputD, iGrad.gpuData()));
  
  CudaM->resetTemp();
  return {{&input}, {&iGrad}};
}
