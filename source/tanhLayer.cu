#include "../header/tanhLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

thread_local hipdnnHandle_t Layer::nnHandle{};
thread_local hipblasHandle_t Layer::blasHandle{};


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

tanhLayer::tanhLayer(){
  TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  TryCuda(hipdnnCreateActivationDescriptor(&actD));
  TryCuda(hipdnnSetActivationDescriptor(actD, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0.0f));
}

tanhLayer::~tanhLayer(){
  TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
  TryCuda(hipdnnDestroyActivationDescriptor(actD));
}

Tensor tanhLayer::forward(const Tensor& T, bool train){
  Tensor output(T.dimensions, TensorLocation::GPU, T.n);
  TryCuda(hipdnnSetTensor4dDescriptor(tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.size, 1, 1, 1));
  TryCuda(hipdnnActivationForward(nnHandle, actD, &mx, tensorD, T.gpuData(), &mn, tensorD, output.gpuData()));
  return output;
}

Tensor tanhLayer::backward(const Tensor& gradient){
  Tensor iGrad(output.dimensions, TensorLocation::GPU, output.n);
  TryCuda(hipdnnActivationBackward(nnHandle, actD, &mx, tensorD, output.gpuData(), tensorD, 
                                  gradient.gpuData(), tensorD, output.gpuData(), &mn, tensorD, iGrad.gpuData()));

  return iGrad; 
}