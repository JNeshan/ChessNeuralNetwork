#include "header/tanhLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnTensorDescriptor_t tensorD;
  hipdnnActivationDescriptor_t actD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
    TryCuda(hipdnnCreateActivationDescriptor(&actD));

  }

  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
    TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  }

  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
    TryCuda(hipdnnDestroyActivationDescriptor(actD));
    TryCuda(hipdnnDestroy(handle));
  };
};

tanhLayer::tanhLayer(){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetActivationDescriptor(CudaM->actD, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0.0f));
}

tanhLayer::~tanhLayer(){
  delete CudaM;
}

std::pair<Tensor, std::unique_ptr<ForwardCache>> tanhLayer::forward(const Tensor& T){
  Tensor output(T.dimensions, TensorLocation::GPU, T.n);
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.size, 1, 1, 1));
  TryCuda(hipdnnActivationForward(CudaM->handle, CudaM->actD, &mx, CudaM->tensorD, T.gpuData(), &mn, CudaM->tensorD, output.gpuData()));
  ForwardCache(output);
  return output;
}

std::pair<Tensor, std::unique_ptr<BackwardCache>> tanhLayer::backward(const Tensor& gradient, const ForwardCache& fCache){
  Tensor iGrad(dimensions, TensorLocation::GPU, n);
  TryCuda(hipdnnActivationBackward(CudaM->handle, CudaM->actD, &mx, CudaM->outputD, output.gpuData(), CudaM->outputD, 
                                  gradient.gpuData(), CudaM->inputD, output.gpuData(), &mn, CudaM->inputD, iGrad.gpuData()));

  CudaM->resetTemp();
  return iGrad; 
}