#include "hip/hip_runtime.h"
#include "header/tanhLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaMembers{
  hipdnnHandle_t handle;
  hipdnnTensorDescriptor_t inputD, outputD;
  hipdnnActivationDescriptor_t actD;

  CudaMembers(){
    TryCuda(hipdnnCreate(&handle));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
    TryCuda(hipdnnCreateActivationDescriptor(&actD));

  }

  void resetTemp(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnCreateTensorDescriptor(&inputD));
    TryCuda(hipdnnCreateTensorDescriptor(&outputD));
  }

  ~CudaMembers(){
    TryCuda(hipdnnDestroyTensorDescriptor(inputD));
    TryCuda(hipdnnDestroyTensorDescriptor(outputD));
    TryCuda(hipdnnDestroyActivationDescriptor(actD));
    TryCuda(hipdnnDestroy(handle));
  };
};

tanhLayer::tanhLayer(){
  CudaM = new CudaMembers();
  TryCuda(hipdnnSetActivationDescriptor(CudaM->actD, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0.0f));

}

tanhLayer::~tanhLayer(){
  delete CudaM;
}

Tensor tanhLayer::forward(const Tensor& T){
  output = Tensor({T.dimensions[0], 1}, TensorLocation::GPU);
  input = Tensor(T);

  if(T.size != output.size){
    throw("Bad tanh input");
  }

  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], 1, 1, 1));
  TryCuda(hipdnnSetTensor4dDescriptor(CudaM->outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], 1, 1, 1));
  TryCuda(hipdnnActivationForward(CudaM->handle, CudaM->actD, &mx, CudaM->inputD, input.gpuData(), &mn, CudaM->outputD, output.gpuData()));
  return output;
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> tanhLayer::backward(const Tensor& gradient){
  iGrad = Tensor(input.dimensions);
  TryCuda(hipdnnActivationBackward(CudaM->handle, CudaM->actD, &mx, CudaM->outputD, output.gpuData(), CudaM->outputD, 
                                  gradient.gpuData(), CudaM->inputD, output.gpuData(), &mn, CudaM->inputD, iGrad.gpuData()));

  CudaM->resetTemp();
  return {{&input}, {&iGrad}}; 
}