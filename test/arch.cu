#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;
int main(){
  int count = 0;
  hipError_t err = hipGetDeviceCount(&count);
  if(err != hipSuccess){
    cout<<"fail 1"<<endl;
    return 1;
  }

  if(!count){
    cout<<"No device"<<endl;
    return 1;
  }

  hipDeviceProp_t deviceProp;
  cudaGetDeviceProperties_v2(&deviceProp, 0);
  cout<<deviceProp.major<<"."<<deviceProp.minor<<endl;
  return 0;
}