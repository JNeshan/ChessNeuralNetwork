#include "hip/hip_runtime.h"
#include "../header/convolutionLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>
#include <chrono>
__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void tensorDesc(hipdnnTensorDescriptor_t& desc, const Tensor& T){
  TryCuda(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], T.dimensions[1], T.dimensions[2], T.dimensions[3]));
}

/*
bias and filter members
hipdnnTensorDescriptor_t inputD, outputD; hipdnnConvolutionDescriptor_t convoD;
hipdnnFilterDescriptor_t filterD; hipdnnTensorDescriptor_t biasD;
*/

ConvolutionLayer::ConvolutionLayer(const int fC, const int iC, const int fH, const int fW, const int pad) : forw(false), back(false), bias({1, fC}, TensorLocation::GPU), filters({fC, iC, fH, fW}, TensorLocation::GPU), fGrad({fC, iC, fH, fW}, TensorLocation::GPU), bGrad({1, fC}, TensorLocation::GPU), padding(pad){
  TryCuda(hipdnnCreateTensorDescriptor(&inputD));
  TryCuda(hipdnnCreateTensorDescriptor(&outputD));
  TryCuda(hipdnnCreateTensorDescriptor(&biasD));
  TryCuda(hipdnnCreateFilterDescriptor(&filterD));
  TryCuda(hipdnnCreateConvolutionDescriptor(&convoD));
  TryCuda(hipdnnCreateActivationDescriptor(&this->actD));
  TryCuda(hipdnnSetFilter4dDescriptor(filterD, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fC, iC, fH, fW));
  TryCuda(hipdnnSetTensor4dDescriptor(biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, fC, 1, 1));
  TryCuda(hipdnnSetConvolution2dDescriptor(convoD, padding, padding, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  TryCuda(hipdnnSetConvolutionMathType(this->convoD, HIPDNN_TENSOR_OP_MATH));
  TryCuda(hipdnnSetActivationDescriptor(this->actD, HIPDNN_ACTIVATION_PATHTRU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  this->convoAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  this->wsPtrF = nullptr;
  this->wsPtrB = nullptr;
}

ConvolutionLayer::ConvolutionLayer(const ConvolutionLayer& lay) : padding(lay.padding), filters(lay.filters), bias(lay.bias), fGrad(lay.fGrad), bGrad(lay.bGrad){
  const auto [k, c, h, w] = std::tie(lay.filters.dimensions[0], lay.filters.dimensions[1], lay.filters.dimensions[2], lay.filters.dimensions[3]);
  TryCuda(hipdnnCreateTensorDescriptor(&this->inputD));
  TryCuda(hipdnnCreateTensorDescriptor(&this->outputD));
  TryCuda(hipdnnCreateTensorDescriptor(&this->biasD));
  TryCuda(hipdnnCreateFilterDescriptor(&this->filterD));
  TryCuda(hipdnnCreateConvolutionDescriptor(&this->convoD));
  TryCuda(hipdnnCreateActivationDescriptor(&this->actD));
  TryCuda(hipdnnSetFilter4dDescriptor(this->filterD, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, h, w));
  TryCuda(hipdnnSetTensor4dDescriptor(this->biasD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, k, 1, 1));
  TryCuda(hipdnnSetConvolution2dDescriptor(this->convoD, this->padding, this->padding, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  TryCuda(hipdnnSetConvolutionMathType(this->convoD, HIPDNN_TENSOR_OP_MATH));
  TryCuda(hipdnnSetActivationDescriptor(this->actD, HIPDNN_ACTIVATION_PATHTRU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

  this->convoAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  this->forw = false;
  this->back = false;
  this->wsSizeF = 0;
  this->wsSizeB = 0;
  this->wsPtrF = nullptr;
  this->wsPtrB = nullptr;
}

ConvolutionLayer::~ConvolutionLayer(){
  TryCuda(hipdnnDestroyFilterDescriptor(this->filterD));
  TryCuda(hipdnnDestroyTensorDescriptor(this->inputD));
  TryCuda(hipdnnDestroyTensorDescriptor(this->outputD));
  TryCuda(hipdnnDestroyTensorDescriptor(this->biasD));
  TryCuda(hipdnnDestroyConvolutionDescriptor(this->convoD));
  if(this->wsPtrF != nullptr){ //frees the workspace if it was used
    TryCuda(hipFree(this->wsPtrF));
  }
  if(this->wsPtrB != nullptr){
    TryCuda(hipFree(this->wsPtrB));
  }
}


std::unique_ptr<Layer> ConvolutionLayer::clone(){
  return(std::make_unique<ConvolutionLayer>(*this));
}

Tensor ConvolutionLayer::forward(Tensor& T, bool train){
  auto start = std::chrono::steady_clock::now();
  auto elapsed = std::chrono::steady_clock::now() - start;
  int n = T.dimensions[0], c = T.dimensions[1], h = T.dimensions[2], w = T.dimensions[3];
  if(filters.dimensions[1] != T.dimensions[1]){
    throw std::runtime_error("Incorrect input channels for convolutional layer");
  }
  TryCuda(hipdnnSetTensor4dDescriptor(inputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w)); //input
  TryCuda(hipdnnGetConvolution2dForwardOutputDim(convoD, inputD, filterD, &n, &c, &h, &w)); //calculates the dimension sizes that the output will have
  TryCuda(hipdnnSetTensor4dDescriptor(outputD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w)); //output

  
  Tensor output({n, c, h, w}, TensorLocation::GPU); //readies return tensor
  
  //variables for convolution algorithm and workspace memory
  //void* workspace = nullptr;
  
  if(!this->forw){ //very time consuming to get the optimal algorithm and details so it only does it once, could replace with a map since it'll always be relative to batch size
    TryCuda(hipdnnGetConvolutionForwardWorkspaceSize(this->nnHandle, this->inputD, this->filterD, 
                                                  this->convoD, this->outputD, this->convoAlgo, &this->wsSizeF));
    
    if(this->wsSizeF > 0){ //allocates necessary workspace space if any
      TryCuda(hipMalloc((void**)&this->wsPtrF, this->wsSizeF));
    }

    this->forw = true;
  }

  //performs convolution
  //hipDeviceSynchronize();
  //auto startL = std::chrono::steady_clock::now();
  TryCuda(cudnnConvolutionBiasActivationForward(this->nnHandle, &mx, this->inputD, T.gpuData(), this->filterD, this->filters.gpuData(), this->convoD, this->convoAlgo,
                                                this->wsPtrF, this->wsSizeF, &mn, this->outputD, output.gpuData(), this->biasD, this->bias.gpuData(), this->actD,
                                                this->outputD, output.gpuData()));
  //hipDeviceSynchronize();
  //elapsed = std::chrono::steady_clock::now() - startL;
  ////std::cout<<std::string("Time in convolution op: ") + std::to_string(std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count())<<std::endl;  
  

  if(train){
    this->input = std::move(T);
  }
  elapsed = std::chrono::steady_clock::now() - start;
  //std::cout<<std::string("Time in convolution: ") + std::to_string(std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count())<<std::endl;  
  return std::move(output);
}

Tensor ConvolutionLayer::backward(Tensor& gradient){
  auto start = std::chrono::steady_clock::now();
  //Tensor iGrad = Tensor(this->input.dimensions, TensorLocation::GPU);
  
  //initializing gradient tensors and descriptor parameters
  
  //bad naming, first two for filter, last two for the returned gradient
  
  //applies back propagation through the convolutions bias

  if(!this->back){
    int algoCount = 0;
    size_t wsSizeX = 0, wsSizeY;
    hipdnnConvolutionBwdFilterAlgoPerf_t potential;
    hipdnnConvolutionBwdDataAlgoPerf_t dataPot;
    TryCuda(hipdnnFindConvolutionBackwardDataAlgorithm(this->nnHandle, this->filterD, this->outputD, this->convoD, this->inputD, 1, &algoCount, &dataPot));
    TryCuda(hipdnnFindConvolutionBackwardFilterAlgorithm(this->nnHandle, this->inputD, this->outputD, this->convoD, this->filterD, 1, &algoCount, &potential));
    this->backDataAlgo = dataPot.algo;
    this->backFilterAlgo = potential.algo;
    TryCuda(hipdnnGetConvolutionBackwardDataWorkspaceSize(this->nnHandle, this->filterD, this->outputD, this->convoD, this->inputD, this->backDataAlgo, &wsSizeX));
    TryCuda(hipdnnGetConvolutionBackwardFilterWorkspaceSize(this->nnHandle, this->inputD, this->outputD, this->convoD, this->filterD, this->backFilterAlgo, &wsSizeY));
    
    
    this->wsSizeB = std::max(wsSizeX, wsSizeY);
    if(this->wsSizeB > 0){ //allocates space for workspace if necessary, uses the higher space requirement so both can use the same allocation on their turn
      TryCuda(hipMalloc((void**)&this->wsPtrB, this->wsSizeB));
    }
    this->back = true;
  }
  TryCuda(hipdnnConvolutionBackwardBias(this->nnHandle, &mx, this->outputD, gradient.gpuData(), &mn, this->biasD, this->bGrad.gpuData()));

  TryCuda(hipdnnConvolutionBackwardFilter(this->nnHandle, &mx, this->inputD, this->input.gpuData(), this->outputD, gradient.gpuData(), 
                                        this->convoD, this->backFilterAlgo, this->wsPtrB, this->wsSizeB, &mn, this->filterD, fGrad.gpuData()));

  TryCuda(hipdnnConvolutionBackwardData(this->nnHandle, &mx, this->filterD, this->filters.gpuData(), this->outputD, 
                                      gradient.gpuData(), this->convoD, this->backDataAlgo, this->wsPtrB, this->wsSizeB, &mn, 
                                      this->inputD, this->input.gpuData()));

  //TryCuda(hipdnnConvolutionBackwardData(nnHandle, &mx, filterD, filters.gpuData(), outputD, 
  //                                    gradient.gpuData(), convoD, dataAlgo, workspace, wsSize, &mn, 
  //                                    inputD, iGrad.gpuData()));
  auto elapsed = std::chrono::steady_clock::now() - start;
  //std::cout<<std::string("Time in convolution back: ") + std::to_string(std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count())<<std::endl;  
  
  return std::move(this->input);
}

void ConvolutionLayer::genTensorData(){
  Generator::tGen(bias);
  Generator::tGen(filters);
}

void ConvolutionLayer::loadTensor(std::ifstream& iF){
  if(!iF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }

  filters.readBinary(iF);
  bias.readBinary(iF);
  filters.gpuSend();
  bias.gpuSend();
}

void ConvolutionLayer::saveTensor(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }

  filters.writeBinary(oF);
  bias.writeBinary(oF);
  filters.gpuSend();
  bias.gpuSend();
}

void ConvolutionLayer::cleanSave(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }
  oF << "Convolutional Layer Tensors:\nFilters ";
  filters.writeTensor(oF);
  oF<<"Bias ";
  bias.writeTensor(oF);
  filters.gpuSend();
  bias.gpuSend();
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> ConvolutionLayer::getLearningData(){
  return {{&filters, &bias}, {&fGrad, &bGrad}};
}

