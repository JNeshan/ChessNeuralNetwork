#include "hip/hip_runtime.h"
#include "../header/denseLayer.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>

//should be done

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipblasStatus_t err){
  if(err != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "cuBLAS Error in %s at line %d: %s\n", __FILE__, __LINE__, cublasGetStatusString(err));
      exit(EXIT_FAILURE);
  }
}



__global__ void biasAddKernel(const float* bias, float* out, const int n, const int m){
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n * m){
    out[idx] += bias[idx % n];
  }
}

__global__ void bGradKernel(const float* grad, float* out, const int m, const int n){
  extern __shared__ float shared[]; //initializes space for shared memory
  const int colIdx = blockIdx.x; //column index the thread block works on
  const int thId = threadIdx.x; //threads relative id to its block
  shared[thId] = 0.0f; //initializing shared memory values
  for(int row = thId; row < m; row += blockDim.x){ //adding gradient values to shared memory
    shared[thId] += grad[row * n + colIdx];
  }
  __syncthreads(); //waits for all threads to finish
  //essentially starts by cutting off half the active threads, only the lower half index of each block starts up to thread th0, 
  //then each active thread adds the value k indices away from their own, where k is 1 plus the index of th0.
  //This then continues to merge down each column until the final two indices containing the sums of their halfs of the vector merge for that
  //rows gradient
  for(int str = blockDim.x / 2; str > 0; str >>= 1){ //summing each column, applying half to the other half each time
    if(thId < str){ //indicates which threads are still allowed
      shared[thId] += shared[thId + str];
    }
    __syncthreads(); //waits to sync each iteration
  }

  if(thId == 0){ //the last thread sets the columns value
    out[colIdx] = shared[0]; 
  }
}

DenseLayer::DenseLayer(const int f, const int n) : weight({f, n}, TensorLocation::GPU), bias({1, n}, TensorLocation::GPU), wGrad({f, n}, TensorLocation::GPU), bGrad({1, n}, TensorLocation::GPU), input(){}

DenseLayer::DenseLayer(const DenseLayer& lay) : weight(lay.weight), bias(lay.bias), wGrad(lay.wGrad.dimensions, TensorLocation::GPU, lay.wGrad.n), bGrad(lay.bGrad.dimensions, TensorLocation::GPU, lay.bGrad.n), input(){
  
}

DenseLayer::~DenseLayer(){
  
}

std::unique_ptr<Layer> DenseLayer::clone(){
  return(std::make_unique<DenseLayer>(*this));
}

Tensor DenseLayer::forward(Tensor& T, bool train){
  auto start = std::chrono::steady_clock::now();
  if(T.n != 2){
    throw std::runtime_error("Wrong dimensional tensor for dense layer.");
  }
  if(T.dimensions[1] != weight.dimensions[0]){
    throw std::runtime_error("Weight and input tensor dimensions incompatible for multiplication");
  }

  Tensor output({T.dimensions[0], weight.dimensions[1]}, TensorLocation::GPU);

  TryCuda(hipblasSgemm(blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, weight.dimensions[1], T.dimensions[0], T.dimensions[1],
             &mx, weight.gpuData(), weight.dimensions[1], T.gpuData(), T.dimensions[1], &mn, output.gpuData(), output.dimensions[1]));
  const int thCount = 256, m = (output.size + thCount - 1) / thCount;
  dim3 blockDim(thCount);
  dim3 gridDim(m);
  biasAddKernel<<<gridDim, blockDim>>>(bias.gpuData(), output.gpuData(), output.dimensions[0], output.dimensions[1]);
  auto elapsed = std::chrono::steady_clock::now() - start;
  if(train){
    this->input = std::move(T);
  }

  ////std::cout<<std::string("Time in dense: ") + std::to_string(std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count())<<std::endl;  
  return output;
}

Tensor DenseLayer::backward(Tensor& gradient){
  
  Tensor iGrad(input.dimensions, TensorLocation::GPU, input.n);

  /*
  inp: n x iF
  wgt: iF x oF, oF x iF
  grad: n x oF
  grad^T x inp = wgt^T : (oF x n) x (n x iF) = (oF x iF)
  */

  //calculates the input gradient
  TryCuda(hipblasSgemm(blasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, weight.dimensions[0], gradient.dimensions[0], gradient.dimensions[1], &mx,
                        weight.gpuData(), weight.dimensions[1], gradient.gpuData(), gradient.dimensions[1], &mn, iGrad.gpuData(), iGrad.dimensions[1])); 

  //calculates weight gradient 

  TryCuda(hipblasSgemm(blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, gradient.dimensions[1], input.dimensions[1], input.dimensions[0],
                        &mx, gradient.gpuData(), gradient.dimensions[1], input.gpuData(), input.dimensions[1], &mn, wGrad.gpuData(), wGrad.dimensions[1]));

  const int thCount = 256; //threads per block
  dim3 gridDim(gradient.dimensions[1]);
  dim3 blockDim(thCount); //one dimensional block of th threads
  size_t shrMemSize = thCount * sizeof(float); //size in memory of each block
  bGradKernel<<<gridDim, blockDim, shrMemSize>>>(gradient.gpuData(), bGrad.gpuData(), gradient.dimensions[0], gradient.dimensions[1]);
  return iGrad;
}

void DenseLayer::genTensorData(){
  Generator::tGen(bias);
  Generator::tGen(weight);
}

void DenseLayer::loadTensor(std::ifstream& iF){
  if(!iF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }

  weight.readBinary(iF);
  bias.readBinary(iF);
  weight.gpuSend();
  bias.gpuSend();
}

void DenseLayer::saveTensor(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }

  weight.writeBinary(oF);
  bias.writeBinary(oF);
  weight.gpuSend();
  bias.gpuSend();
}

void DenseLayer::cleanSave(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }
  oF << "Dense Layer Tensors:\nWeights ";
  weight.writeTensor(oF);
  oF << "Bias ";
  bias.writeTensor(oF);
  weight.gpuSend();
  bias.gpuSend();
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> DenseLayer::getLearningData(){
  return {{&weight, &bias}, {&wGrad, &bGrad}};
}