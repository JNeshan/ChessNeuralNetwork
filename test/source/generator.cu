#include "../header/generator.h"

__inline__ void TryCuda(hiprandStatus_t err){
  if(err != HIPRAND_STATUS_SUCCESS){
    fprintf(stderr, "hiprand Error in %s at line %d: %s\n", __FILE__, __LINE__, (char)('0' + err));
      exit(EXIT_FAILURE);
  }
}

__inline__ hiprandGenerator_t createCurand(){
  hiprandGenerator_t gen;
  TryCuda(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  TryCuda(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
  TryCuda(hiprandSetGeneratorOffset(gen, 1));
  TryCuda(hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_DEFAULT));
  return gen;
}



hiprandGenerator_t Generator::cGen = createCurand();

Generator::Generator(){
  int a = 0;
}

void Generator::tGen(const Tensor& T){
  float* output = T.gpuData();
  TryCuda(hiprandGenerateUniform(cGen, output, T.size));
  return;
}

void Generator::dGen(const int s, float* data){
  TryCuda(hiprandGenerateUniform(cGen, data, s));
}