#include "hip/hip_runtime.h"
#include "../header/generator.h"

__inline__ void TryCuda(hiprandStatus_t err){
  if(err != HIPRAND_STATUS_SUCCESS){
    fprintf(stderr, "hiprand Error in %s at line %d: %s\n", __FILE__, __LINE__, (char)('0' + err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "hiprand Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ hiprandGenerator_t createCurand(){
  hiprandGenerator_t gen;
  TryCuda(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  TryCuda(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
  TryCuda(hiprandSetGeneratorOffset(gen, 1));
  TryCuda(hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_DEFAULT));
  return gen;
}

__global__ void ascendKernel(const int s, float* data){
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < s){
    data[idx] = idx;
  }
}

__global__ void valKernel(const int s, const int v, float* data){
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < s){
    data[idx] = v;
  }
}

__global__ void copyKernel(const int s, const float* r, float* data){
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < s){
    data[idx] = r[idx];
  }
}



hiprandGenerator_t Generator::cGen = createCurand();

Generator::Generator(){
  int a = 0;
}

void Generator::tGen(const Tensor& T){
  float* output = T.gpuData();
  TryCuda(hiprandGenerateUniform(cGen, output, T.size));
  return;
}

void Generator::dGen(const int s, float* data){
  TryCuda(hiprandGenerateUniform(cGen, data, s));
}

void Generator::aGen(const int s, float* data){
  const int thCount = 256, m = (s + thCount - 1) / thCount;
  dim3 blockDim(thCount);
  dim3 gridDim(m);
  ascendKernel<<<gridDim, blockDim>>>(s, data);
  TryCuda(hipGetLastError());
  TryCuda(hipDeviceSynchronize());
}

void Generator::vGen(const int s, const int v, float* data){
  const int thCount = 256, m = (s + thCount - 1) / thCount;
  dim3 blockDim(thCount);
  dim3 gridDim(m);
  valKernel<<<gridDim, blockDim>>>(s, v, data);
  TryCuda(hipGetLastError());
  TryCuda(hipDeviceSynchronize());
}

void Generator::copy(const int s, const float* r, float* data){
  const int thCount = 256, m = (s + thCount - 1) / thCount;
  dim3 blockDim(thCount);
  dim3 gridDim(m);
  copyKernel<<<gridDim, blockDim>>>(s, r, data);
  TryCuda(hipGetLastError());
  TryCuda(hipDeviceSynchronize());
}