#include "../header/reluLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

ReLULayer::ReLULayer(){
  TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  TryCuda(hipdnnCreateActivationDescriptor(&reLU));  
  TryCuda(hipdnnSetActivationDescriptor(reLU, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0));
}

ReLULayer::ReLULayer(const ReLULayer& lay){
  TryCuda(hipdnnCreateTensorDescriptor(&this->tensorD));
  TryCuda(hipdnnCreateActivationDescriptor(&this->reLU));
  TryCuda(hipdnnSetActivationDescriptor(this->reLU, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0));
}

std::unique_ptr<Layer> ReLULayer::clone(){
  return(std::make_unique<ReLULayer>(*this));
}


Tensor ReLULayer::forward(Tensor& T, bool train){
  if(train){
    this->input = T;
  }
  auto start = std::chrono::steady_clock::now();

  auto elapsed = std::chrono::steady_clock::now() - start;
  TryCuda(hipdnnSetTensor4dDescriptor(tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.size, 1, 1, 1));
  TryCuda(hipdnnActivationForward(nnHandle, reLU, &mx, tensorD, T.gpuData(), &mn, tensorD, T.gpuData()));
  
  elapsed = std::chrono::steady_clock::now() - start;
  ////std::cout<<std::string("Time in relu: ") + std::to_string(std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count())<<std::endl;  
  return T;
}

Tensor ReLULayer::backward(Tensor& gradient){
  //Tensor iGrad(input.dimensions, TensorLocation::GPU, input.n);
  TryCuda(hipdnnActivationBackward(nnHandle, reLU, &mx, tensorD, gradient.gpuData(), tensorD, gradient.gpuData(), tensorD, input.gpuData(), &mn, tensorD, gradient.gpuData()));  
  return gradient;
}

void ReLULayer::saveTensor(std::ofstream& oF){
  return;
}
void ReLULayer::genTensorData(){
  return;
}
void ReLULayer::loadTensor(std::ifstream& iF){
  return;
}

void ReLULayer::cleanSave(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }
  input.writeTensor(oF);
}


ReLULayer::~ReLULayer(){
  TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
  TryCuda(hipdnnDestroyActivationDescriptor(reLU));
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> ReLULayer::getLearningData(){
  return {};
}