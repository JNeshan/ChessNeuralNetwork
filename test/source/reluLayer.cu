#include "../header/reluLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

ReLULayer::ReLULayer(){
  TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  TryCuda(hipdnnCreateActivationDescriptor(&reLU));  
  TryCuda(hipdnnSetActivationDescriptor(reLU, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0));


}

Tensor ReLULayer::forward(const Tensor& T, bool train){
  
  Tensor output(T.dimensions, TensorLocation::GPU, T.n);
  TryCuda(hipdnnSetTensor4dDescriptor(tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.size, 1, 1, 1));
  TryCuda(hipdnnActivationForward(nnHandle, reLU, &mx, tensorD, T.gpuData(), &mn, tensorD, output.gpuData()));
  return output;
}

Tensor ReLULayer::backward(const Tensor& gradient){
  Tensor iGrad(input.dimensions, TensorLocation::GPU, input.n);
  TryCuda(hipdnnActivationBackward(nnHandle, reLU, &mx, tensorD, gradient.gpuData(), tensorD, gradient.gpuData(), tensorD, input.gpuData(), &mn, tensorD, iGrad.gpuData()));  
  return iGrad;
}

void ReLULayer::saveTensor(std::ofstream& oF){
  return;
}
void ReLULayer::genTensorData(){
  return;
}
void ReLULayer::loadTensor(std::ifstream& iF){
  return;
}

void ReLULayer::cleanSave(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }
  input.writeTensor(oF);
}


ReLULayer::~ReLULayer(){
  TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
  TryCuda(hipdnnDestroyActivationDescriptor(reLU));
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> ReLULayer::getLearningData(){
  return {};
}