#include "../header/softmaxLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>
#include <string>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

struct CudaCache{
  hipdnnTensorDescriptor_t tensorD;
  CudaCache(){
    TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  }
  ~CudaCache(){
    TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
  }
};

SoftmaxLayer::SoftmaxLayer() : outFeat(4672){
  TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
}

SoftmaxLayer::~SoftmaxLayer(){
  TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
}

Tensor SoftmaxLayer::forward(const Tensor& T){

  if(T.n != 2){ //input must be 2 dimensional
    throw("Softmax input invalid n = " + std::to_string(T.n));
  }
  Tensor({T.dimensions[0], outFeat}, TensorLocation::GPU);
  output = Tensor({T.dimensions[0], outFeat}, TensorLocation::GPU); //storing output for back
  if(output.size != T.size){ //check to ensure the matrices are the same size (also means 2nd dimensions are equal)
    throw("Bad softmax input"); 
  }
  TryCuda(hipdnnSetTensor4dDescriptor(tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], T.dimensions[1], 1, 1)); //input descriptor
  TryCuda(hipdnnSoftmaxForward(nnHandle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, tensorD, T.gpuData(), &mn, tensorD, output.gpuData()));
  return output;
}

Tensor SoftmaxLayer::backward(const Tensor& gradient){
  if(gradient.n != 2 || gradient.size < output.size){
    throw("Softmax recieved bad gradient or recorded faulty output");
  }
  Tensor iGrad(output.dimensions, TensorLocation::GPU, output.n);
  TryCuda(hipdnnSoftmaxBackward(nnHandle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, tensorD, 
                              output.gpuData(), tensorD, gradient.gpuData(), &mn, tensorD, iGrad.gpuData()));
  return iGrad;
}

void SoftmaxLayer::saveTensor(std::ofstream& oF){
  return;
}
void SoftmaxLayer::genTensorData(){
  return;
}
void SoftmaxLayer::loadTensor(std::ifstream& iF){
  return;
}

void SoftmaxLayer::cleanSave(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }
  oF << "Softmax Layer Tensor:\n";
  output.writeTensor(oF);
}