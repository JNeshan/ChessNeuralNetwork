#include "../header/softmaxLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>
#include <string>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}


SoftmaxLayer::SoftmaxLayer() : outFeat(4672){
  TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
}

SoftmaxLayer::SoftmaxLayer(const SoftmaxLayer& lay) : outFeat(lay.outFeat){
  TryCuda(hipdnnCreateTensorDescriptor(&this->tensorD));
}

SoftmaxLayer::~SoftmaxLayer(){
  TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
}

std::unique_ptr<Layer> SoftmaxLayer::clone(){
  return(std::make_unique<SoftmaxLayer>(*this));
}

Tensor SoftmaxLayer::forward(Tensor& T, bool train){

  if(T.n != 2){ //input must be 2 dimensional
    throw std::runtime_error("Softmax input invalid n = " + std::to_string(T.n));
  }
  this->output = Tensor({T.dimensions[0], outFeat}, TensorLocation::GPU); //storing output for back
  if(output.size != T.size){ //check to ensure the matrices are the same size (also means 2nd dimensions are equal)
    throw std::runtime_error("Bad softmax input"); 
  }
  TryCuda(hipdnnSetTensor4dDescriptor(tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.dimensions[0], T.dimensions[1], 1, 1)); //input descriptor
  TryCuda(hipdnnSoftmaxForward(nnHandle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, tensorD, T.gpuData(), &mn, tensorD, output.gpuData()));
  return output;
}

Tensor SoftmaxLayer::backward(Tensor& gradient){
  if(gradient.n != 2 || gradient.size < output.size){
    throw std::runtime_error("Softmax recieved bad gradient or recorded faulty output");
  }
  Tensor iGrad(output.dimensions, TensorLocation::GPU, output.n);
  TryCuda(hipdnnSoftmaxBackward(nnHandle, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &mx, tensorD, 
                              output.gpuData(), tensorD, gradient.gpuData(), &mn, tensorD, iGrad.gpuData()));
  return iGrad;
}

void SoftmaxLayer::saveTensor(std::ofstream& oF){
  return;
}
void SoftmaxLayer::genTensorData(){
  return;
}
void SoftmaxLayer::loadTensor(std::ifstream& iF){
  return;
}

void SoftmaxLayer::cleanSave(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }
  oF << "Softmax Layer Tensor:\n";
  output.writeTensor(oF);
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> SoftmaxLayer::getLearningData(){
  return {};
}