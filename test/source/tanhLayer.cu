#include "hip/hip_runtime.h"
#include "../header/tanhLayer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdexcept>


__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ void TryCuda(hipdnnStatus_t err){
  if(err != HIPDNN_STATUS_SUCCESS){
    fprintf(stderr, "CUDNN Error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

tanhLayer::tanhLayer(){
  TryCuda(hipdnnCreateTensorDescriptor(&tensorD));
  TryCuda(hipdnnCreateActivationDescriptor(&actD));
  TryCuda(hipdnnSetActivationDescriptor(actD, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0.0f));
}

tanhLayer::tanhLayer(const tanhLayer& lay){
  TryCuda(hipdnnCreateTensorDescriptor(&this->tensorD));
  TryCuda(hipdnnCreateActivationDescriptor(&this->actD));
  TryCuda(hipdnnSetActivationDescriptor(this->actD, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0.0f));
}

std::unique_ptr<Layer> tanhLayer::clone(){
  return(std::make_unique<tanhLayer>(*this));
}

tanhLayer::~tanhLayer(){
  TryCuda(hipdnnDestroyTensorDescriptor(tensorD));
  TryCuda(hipdnnDestroyActivationDescriptor(actD));
}

Tensor tanhLayer::forward(Tensor& T, bool train){
  
  TryCuda(hipdnnSetTensor4dDescriptor(tensorD, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, T.size, 1, 1, 1));
  TryCuda(hipdnnActivationForward(nnHandle, actD, &mx, tensorD, T.gpuData(), &mn, tensorD, T.gpuData()));
  if(train){
    this->output = T;
  }
  return std::move(T);
}

Tensor tanhLayer::backward(Tensor& gradient){
  //Tensor iGrad(output.dimensions, TensorLocation::GPU, output.n);
  TryCuda(hipdnnActivationBackward(nnHandle, actD, &mx, tensorD, output.gpuData(), tensorD, 
                                  gradient.gpuData(), tensorD, output.gpuData(), &mn, tensorD, gradient.gpuData()));

  return gradient; 
}

void tanhLayer::saveTensor(std::ofstream& oF){
  return;
}
void tanhLayer::genTensorData(){
  return;
}
void tanhLayer::loadTensor(std::ifstream& iF){
  return;
}

void tanhLayer::cleanSave(std::ofstream& oF){
  if(!oF.is_open()){
    std::cout<<"File not open"<<std::endl;
    return;
  }
  oF <<"tanh Layer Tensor:\n";
  output.writeTensor(oF);
}

std::pair<std::vector<Tensor*>, std::vector<Tensor*>> tanhLayer::getLearningData(){
  return {};
}