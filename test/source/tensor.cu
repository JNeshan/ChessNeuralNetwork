#include "hip/hip_runtime.h"
#include <stdexcept>
#include <vector>
#include "../header/tensor.h"
#include "hip/hip_runtime.h"
#include <string>
#include <sstream>
#include <iomanip>

__inline__ void TryCuda(hipError_t err){
  if(err != hipSuccess){
    fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

__inline__ std::string formatFloat(float f, int width, int precision){
  std::ostringstream oss;
  oss << std::fixed << std::setprecision(precision)<<f;
  std::string out = oss.str();

  if(out.length() < width){
    out = std::string(width - out.length(), ' ') + out;
  }
  else if(out.length() > width){
    out = out.substr(0, width);
  }
  return out;
}

__global__ void AddKernel(const float* A, const float* B, float* out, const int n, const int m){
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n * m){
    if(A == out){
      out[idx] += B[idx];
    }
    else{
      out[idx] = B[idx] + A[idx];
    }
  }
}

Tensor::~Tensor() {
  if(this->size <= 0 || this->data == nullptr){ //blank tensor
    return;
  }
  if(this->device == TensorLocation::GPU){ //lazy delete
    TryCuda(hipFree(this->data));
    this->size = -1;
    return;
  }

  delete[] this->data;
}

Tensor::Tensor() : size(-1), device(TensorLocation::CPU), data(nullptr), dimensions() {}

Tensor::Tensor(const std::vector<int>& dim, const TensorLocation loc, int nth) : n(nth), dimensions(dim), device(loc){
  if(dim.size() == 0){
    throw std::invalid_argument("Tensor dimensions empty");
  }
  if(!nth){ //nth defaults to 0 if no dimension count is given so dim can instead relay the proper n as long as it is not padded
    this->n = dim.size();
  }

  this->size = 1;
  for(auto d : dim){
    if(d < 1) throw std::runtime_error("Non-positive dimension size");
    this->size *= d;
  }

  while(this->dimensions.size() < 4 || this->dimensions.size() < n){
    this->dimensions.push_back(1);
  }

  if(loc == TensorLocation::CPU){
    this->device = TensorLocation::CPU;
    this->data = new float[this->size]();
  }
  else{
    device = TensorLocation::GPU;
    TryCuda(hipMalloc((void**)&data, this->size * sizeof(float)));
  }
}

Tensor::Tensor(const Tensor& r){
  this->device = r.device;
  this->dimensions = r.dimensions;
  this->n = r.n;
  this->size = r.size;
  if(r.data != nullptr){
    if(this->device == TensorLocation::GPU){
      TryCuda(hipMalloc((void**)&data, size * sizeof(float)));
      TryCuda(hipMemcpy(this->data, r.data, size * sizeof(float), hipMemcpyDeviceToDevice));
    }
    else{
      data = new float[size];
      memcpy(data, r.data, size * sizeof(float));
    }
  }
}

Tensor::Tensor(Tensor&& r) : dimensions(r.dimensions), size(r.size), data(r.data), device(r.device), n(r.n){
  r.data = nullptr;
}

Tensor& Tensor::operator=(Tensor&& r){
  if(this == &r){
    return *this;
  }

  if(this->data){
    if(this->device == TensorLocation::GPU){
      TryCuda(hipFree(this->data));
    }
    else{
      delete[] this->data;
    }
  }

  this->size = r.size;
  this->device = r.device;
  this->dimensions = r.dimensions;
  this->n = r.n;
  this->data = r.data;
  r.data = nullptr; 
  return *this;
}

Tensor& Tensor::operator=(const Tensor& r){
  if(this == &r){
    return *this;
  }
  
  if(this->data != nullptr){
    if(this->device == TensorLocation::GPU){
      TryCuda(hipFree(data));
    }
    else{
      delete[] this->data;
    }
  }

  this->size = r.size;
  this->device = r.device;
  this->dimensions = r.dimensions;
  this->n = r.n;
  this->data = nullptr;
  if(this->device == TensorLocation::GPU){
    TryCuda(hipMalloc((void**)&data, this->size * sizeof(float)));
    TryCuda(hipMemcpy(data, r.data, this->size * sizeof(float), hipMemcpyDeviceToDevice));
  }
  else{
    this->data = new float[this->size];
    memcpy(this->data, r.data, this->size * sizeof(float));
  }
  return *this;
}

float* Tensor::cpuData() const{
  if(this->device == TensorLocation::GPU){
    throw std::invalid_argument("Data stored on GPU");
  }
  else if(data == nullptr){
    throw std::invalid_argument("Data not initialized");
  }
  return data;
}

float* Tensor::gpuData() const{
  if(this->device == TensorLocation::CPU){
    throw std::invalid_argument("Data stored on CPU");
  }
  else if(data == nullptr){
    throw std::invalid_argument("Data not initialized");
  }
  return data;
}

void Tensor::cpuSend(){
  if(this->device == TensorLocation::CPU){
    //std::cout<<"Already on CPU"<<std::endl;
    return;
  }
  if(this->data == nullptr){
    throw std::invalid_argument("Data has not been allocated");
  }
  if(this->size <= 0){
    this->device == TensorLocation::CPU;
    return;
  }
  float* tmpData = new float[this->size];
  TryCuda(hipMemcpy(tmpData, this->data, this->size * sizeof(float), hipMemcpyDeviceToHost));
  TryCuda(hipFree(data));
  this->data = tmpData;
  tmpData = nullptr;
  this->device = TensorLocation::CPU;
}

void Tensor::gpuSend(){

  if(this->device == TensorLocation::GPU){
   // std::cout<<"Tensor already in GPU memory"<<std::endl;
    return; //already in gpu memory
  }
  if(this->data == nullptr){
    throw std::invalid_argument("Data has not been allocated");
  }
  if(this->size <= 0){
    this->device == TensorLocation::GPU;
    return;
  }
  float* tmpData;
  TryCuda(hipMalloc((void**)&tmpData, size * sizeof(float))); //allocating memory within the GPU
  TryCuda(hipMemcpy(tmpData, data, size * sizeof(float), hipMemcpyHostToDevice));  //copying into GPU memory
  
  delete[] this->data;
  this->data = tmpData;
  tmpData = nullptr;
  this->device = TensorLocation::GPU;
}

void Tensor::reshape(const std::vector<int>& dim, const int nth){
  int s = 1;
  for(auto d : dim){
    if(d < 1){
      throw std::runtime_error("Non-positive dimension rehsape size");
    }
    s *= d;
  }

  if(s != size){ //the new shape must have the same size as the previous to be valid
    throw std::runtime_error("Different reshape size");
  }
  this->n = nth;
  this->dimensions = dim;
  while(this->dimensions.size() < 4){
    this->dimensions.push_back(1);
  }
}

Tensor Tensor::segment(const int n_i){
  try
  {
    if(n_i < 0 || n_i > this->dimensions[0]){
      throw std::runtime_error("Bad batch index to copy");
    }
      std::vector<int> tDim = this->dimensions;
      tDim[0] = 1;
      Tensor T(tDim, this->device, this->n);
      int ind = n_i * (this->size / this->dimensions[0]);
      if(this->device == TensorLocation::CPU){
        
        memcpy(T.cpuData(), this->data + ind, T.size * sizeof(float));
      }
      else{
        TryCuda(hipMemcpy(T.gpuData(), this->data + ind, T.size * sizeof(float), hipMemcpyDeviceToDevice));
      }
    return std::move(T);
  }
  catch(const std::exception& e)
  {
    std::cerr << e.what() << '\n';
  }
  return Tensor();
}
//dont use
Tensor::Tensor(std::vector<Tensor>& t) : dimensions(t[0].dimensions), size(t[0].size), n(t[0].n){
  this->dimensions[0] = t.size();
  this->size *= this->dimensions[0];
}

void Tensor::batchBuild(const std::vector<Tensor*>& t, const TensorLocation loc){
  if(t.size() == 0){
    throw std::runtime_error("Empty batch vector");
  }
  this->dimensions = t[0]->dimensions;
  this->dimensions[0] = t.size();
  this->size = t[0]->size * t.size();
  this->n = t[0]->n;
  this->device = loc;
  if(loc == TensorLocation::CPU){
    this->data = new float[this->size];
    int i = 0;
    for(auto tensor : t){
      auto idx = this->data + (i * tensor->size);
      if(tensor->device == TensorLocation::CPU){
        memcpy(idx, tensor->cpuData(), tensor->size * sizeof(float));
      }
      else{
        TryCuda(hipMemcpy(idx, tensor->gpuData(), tensor->size * sizeof(float), hipMemcpyDeviceToHost));
      }
      i++;
    }
  }
  else{
    TryCuda(hipMalloc((void**)&this->data, this->size * sizeof(float)));
    int i = 0;
    for(auto tensor : t){
      auto idx = this->data + (i * tensor->size);
      auto cudaCpy = tensor->device == TensorLocation::GPU ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
      TryCuda(hipMemcpy(idx, tensor->data, tensor->size * sizeof(float), cudaCpy));
      i++;
    }
  }
}

void Tensor::batchBuild(const std::vector<Tensor>& t, const TensorLocation loc){
  if(t.size() == 0){
    throw std::runtime_error("Empty batch vector");
  }
  this->dimensions = t[0].dimensions;
  this->dimensions[0] = t.size();
  this->size = t[0].size * t.size();
  this->n = t[0].n;
  this->device = loc;
  if(loc == TensorLocation::CPU){
    this->data = new float[this->size];
    int i = 0;
    for(auto& tensor : t){
      auto idx = this->data + (i * tensor.size);
      if(tensor.device == TensorLocation::CPU){
        memcpy(idx, tensor.cpuData(), tensor.size * sizeof(float));
      }
      else{
        TryCuda(hipMemcpy(idx, tensor.gpuData(), tensor.size * sizeof(float), hipMemcpyDeviceToHost));
      }
      i++;
    }
  }
  else{
    TryCuda(hipMalloc((void**)&this->data, this->size * sizeof(float)));
    int i = 0;
    for(auto tensor : t){
      auto idx = this->data + (i * tensor.size);
      auto cudaCpy = tensor.device == TensorLocation::GPU ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
      TryCuda(hipMemcpy(idx, tensor.data, tensor.size * sizeof(float), cudaCpy));
      i++;
    }
  }
}
//appending
void Tensor::writeTensor(std::ofstream& oF){
  int width = 16, precision = 14;
  if(!oF.is_open()){
    std::cout<<"Output stream not open"<<std::endl;
    return;
  }
  if(data == nullptr || size <= 0){
    std::cout<<"Tensor not allocated"<<std::endl;
    return;
  }
  if(device == TensorLocation::GPU){
    this->cpuSend();
  }
  oF << std::to_string(this->n) << "-dimensional tensor " << this->dimensions[0];
  for(int i = 1; i < this->n; i++){
    oF << " x " << this->dimensions[i];
  }
  oF << "\n";
  int x = this->dimensions[0], y = this->size / x;
  for(int i = 0; i < x; i++){
    for(int j = 0; j < y; j++){
      std::string value = formatFloat(data[i*y + j], width, precision);
      oF << value << " ";
    }
    oF << "\n";
  }
  oF << "\n";
}

void Tensor::writeBinary(std::ofstream& oF){
  if(data == nullptr){
    std::cout<<"Tensor not populated"<<std::endl;
    return;
  }
  if(!oF.is_open()){
    std::cout<<"Output not open"<<std::endl;
    return;
  }
  if(this->device == TensorLocation::GPU){
    this->cpuSend();
  }
  oF.write(reinterpret_cast<const char*>(&this->size), sizeof(int));
  oF.write(reinterpret_cast<const char*>(&this->n), sizeof(int));
  oF.write(reinterpret_cast<const char*>(this->dimensions.data()), this->dimensions.size() * sizeof(int));
  oF.write(reinterpret_cast<const char*>(this->data), this->size * sizeof(float));
}

void Tensor::readBinary(std::ifstream& iF){
  if(!iF.is_open()){
    std::cout<<"File not open to read"<<std::endl;
    return;
  }
  if(this->data == nullptr){
    std::cout<<std::endl;
  }
  else if(this->device == TensorLocation::GPU){
    TryCuda(hipFree(data));
  }
  else{
    delete[] data;
  }
  iF.read(reinterpret_cast<char*>(&this->size), sizeof(int));
  iF.read(reinterpret_cast<char*>(&this->n), sizeof(int));
  int d = std::max(4, this->n);
  dimensions = std::vector<int>(d);
  data = new float[this->size];
  iF.read(reinterpret_cast<char*>(this->dimensions.data()), d * sizeof(int));
  iF.read(reinterpret_cast<char*>(this->data), this->size * sizeof(float));
  this->device = TensorLocation::CPU;
}

void Tensor::gpuAdd(Tensor& B){

  if(this->size != B.size){
    throw std::runtime_error("Different sizes for addition");
  }
  this->gpuSend();
  B.gpuSend();
  
  float* bData = B.gpuData();
  float* aData = this->gpuData();

  int n = this->dimensions[0], m = this->size / n, thrdCnt = 256;
  dim3 gridDim((this->size + thrdCnt - 1) / thrdCnt), blockDim(thrdCnt);

  AddKernel<<<gridDim, blockDim>>>(aData, bData, aData, n, m);
}

float* Tensor::gpuDataForce(){
  this->gpuSend();
  return this->data;
}

float* Tensor::cpuDataForce(){
  this->cpuSend();
  return this->data;
}